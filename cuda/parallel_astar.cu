#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"
#include "parallel_hash.cuh"

struct link {
    slider_state_t node;
    double cost;
    link() {
        slider_state_t node;
        node.zero_idx = DIM_X * DIM_Y;
        link(node, 0);
    }
    link(slider_state_t node, double cost) {
        this->node = node;
        this->cost = cost;
    }
};

__device__ inline bool is_goal(slider_state_t node) {
    for (unsigned char i = 0; i < DIM_X * DIM_Y; ++i) {
        if (node.board[i] != i) return false;
    }

    return true;
}

__device__ inline link* get_next(slider_state_t node) {
    for (unsigned char move = UP; move <= RIGHT; ++move) {
        slider_state_t next_state = state;
        unsigned char new_zero;
        switch (move) {
            case DOWN:
                if (state.zero_idx < DIM_X) continue;
                new_zero = state.zero_idx - DIM_X;
                break;
            case UP:
                if (state.zero_idx >= DIM_X * DIM_Y - DIM_X) continue;
                new_zero = state.zero_idx + DIM_X;
                break;
            case RIGHT:
                if (state.zero_idx % DIM_X == 0) continue;
                new_zero = state.zero_idx - 1;
                break;
            case LEFT:
                if (state.zero_idx % DIM_X == DIM_X - 1) continue;
                new_zero = state.zero_idx + 1;
                break;
        }
        next_state.board[state.zero_idx] = next_state.board[new_zero];
        next_state.board[new_zero] = 0;
        next_state.zero_idx = new_zero;

        unsigned short cost = visited[state].cost + 1;
}

__device__ inline double heuristic(slider_state_t node) {
    return 0;
}


__global__ void parallel_astar_kernel(
    slider_state_t start,
    int num_queues,
    HashTable *closed_set, 
    Node *S, 
    int *best_cost,
    bool *all_queue_empty,
    bool *min_goal_reached
) {
    if (num_queues < 1) {
        throw "We need at least one priority queue";
    }

    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Priority queue based on cost plus heuristic
    heap* pq = new_heap();

    // Array S for the neighbors of the nodes on the frontier (to be deduplicated)
    const uint neighbors_size = 4 * num_queues;

    // Initialize data structures
    Node begin = Node(start, 0, heuristic(start));
    Node init_node_list[] = {begin};
    if (thread_idx == 0) {
        insert_deduplicate(closed_set, init_node_list, 1, pq);
    }

    // If all queues are empty, then failed to find goal state
    atomicAnd(all_queue_empty, is_empty_heap(pq));
    __syncthreads();

    while (!(*all_queue_empty)) {
        if (!is_empty_heap(open_set[i])) {
            element q = peak_heap(pq)[0];
            pop_heap(pq);
            slider_state_t node = q.value;
            if (is_goal(node)) {
                printf("reached a goal state!\n");
                int fnode = q.priority;
                atomicMin(best_cost, fnode);
            } else {
                int cntr = 0;

                for (unsigned char move = UP; move <= RIGHT; ++move) {
                    unsigned short cost = query(closed_set, node).g + 1;

                    slider_state_t next_state = node;
                    unsigned char new_zero;
                    switch (move) {
                        case DOWN:
                            if (state.zero_idx < DIM_X) continue;
                            new_zero = state.zero_idx - DIM_X;
                            break;
                        case UP:
                            if (state.zero_idx >= DIM_X * DIM_Y - DIM_X) continue;
                            new_zero = state.zero_idx + DIM_X;
                            break;
                        case RIGHT:
                            if (state.zero_idx % DIM_X == 0) continue;
                            new_zero = state.zero_idx - 1;
                            break;
                        case LEFT:
                            if (state.zero_idx % DIM_X == DIM_X - 1) continue;
                            new_zero = state.zero_idx + 1;
                            break;
                    }
                    next_state.board[state.zero_idx] = next_state.board[new_zero];
                    next_state.board[new_zero] = 0;
                    next_state.zero_idx = new_zero;

                    Node new_node = Node(next_state, move, node, cost, cost+heuristic(next_state));
                    S[4 * thread_idx + cntr] = new_node;
                    cntr++;
                }
            }
        }

        // return best path if goal is found and there is no
        // element in any of the pq's that are less than current
        // path to goal.
        *min_goal_reached = true;
        __syncthreads();
        if (*best_cost != UINT32_MAX) {
            for (int i = 0; i < 4; i++) {
                Node n = S[i * num_queues + thread_idx];
                // unpopulated entry
                if (n.id.zero_idx == DIM_X * DIM_Y) continue;

                atomicAnd(min_goal_reached, n.f >= *best_cost);
            }

            if (!is_empty_heap(pq) && peak_heap(pq)[0].priority < fm) {
                atomicAnd(min_goal_reached, peak_heap(pq).priority >= *best_cost);
            }

            __syncthreads();
            if (all_less) {
                return;
            }
        }

        // deduplication section
        // run in parallel
        for (size_t i = 0; i < 4; i++) {
            Node n = S[i * num_queues + thread_idx];
            if (query_cost_check(closed_set, n)) {
                S[i * num_queues + thread_idx].id.zero_idx = DIM_X * DIM_Y;
            }
        }
        __syncthreads();

        // insert the remaining nodes in parallel in closed array
        // and priority queues.
        for (int i = 0; i < 4; i++) {
            Node n = S[i * num_queues + thread_idx];
            if (n.id.zero_idx == DIM_X * DIM_Y) continue;

            push_heap(pq, n.id, n.f);

            int z = 0;
            int ind0 = hash_fn1(n.id, closed_set->size);
            int ind1 = hash_fn2(n.id, closed_set->size);

            if (closed_set->table[ind0].id == node_list[i].id || closed_set->table[ind0].id == -1) {
                z = 0;
            } else if (closed_set->table[ind1].id == node_list[i].id || closed_set->table[ind1].id == -1) {
                z = 1;
            }

            bool to_keep = query_cost_check(closed_set, n);
            if (z == 0 && to_keep) {
                Node old = atomicExch(closed_set->table[ind0], n);
                n = old;
            } else if (z == 1 && to_keep) {
                Node old = atomicExch(closed_set->table[ind1], n);
                n = old;    
            }
        }

        *all_queue_empty = true;
        __syncthreads();
        atomicAnd(all_queue_empty, is_empty_heap(pq));
        __syncthreads();
    }
}

void cuda_astar(slider_state_t start, int num_queues, int hash_table_size) {
    // Compute number of blocks and threads per block
    // We use one thread per priority queue
    const int threadsPerBlock = 256;
    const int blocks = (num_queues + threadsPerBlock - 1) / threadsPerBlock;

    // HashTable* closed_set = create_hash_table(hash_table_size);
    Node S[4 * num_queues];
    int best_cost = UINT32_MAX;

    HashTable* device_closed_set;
    Node* device_table;
    Node* device_S;
    int* device_best_cost;
    bool* device_all_queue_empty;
    bool* device_min_goal_reached;

    hipMalloc((void **) &device_closed_set, sizeof(HashTable));
    hipMalloc((void **) &device_table, sizeof(Node) * hash_table_size);
    hipMalloc((void **) &device_S, sizeof(Node) * 4 * num_queues);
    hipMalloc((void **) &device_best_cost, sizeof(int));
    hipMalloc((void **), &device_all_queue_empty, sizeof(bool));
    hipMalloc((void **), &device_min_goal_reached, sizeof(bool));

    // hipMemcpy(device_closed_set, closed_set, sizeof(HashTable), hipMemcpyHostToDevice);
    device_closed_set = create_hash_table(hash_table_size)
    // hipMemcpy(device_table, closed_set->table, sizeof(Node) * hash_table_size, hipMemcpyHostToDevice);
    device_table = device_closed_set->table;
    hipMemcpy(device_S, S, sizeof(Node) * 4 * num_queues, hipMemcpyHostToDevice);
    hipMemcpy(device_best_cost, &best_cost, sizeof(int), hipMemcpyHostToDevice);
    *device_all_queue_empty = true;
    *device_min_goal_reached = true;

    double kernelStartTime = CycleTimer::currentSeconds();
    parallel_astar_kernel<<<blocks, threadsPerBlock>>>(start, num_queues, device_closed_set, device_S, device_best_cost, device_all_queue_empty, device_min_goal_reached);
    hipDeviceSynchronize();

    double kernelEndTime = CycleTimer::currentSeconds();

    // determine the actions using goal_state here
    uarray* backtrack_path = new_uarray(num_queues);
    slider_state_t curr;
    for (int i = 0; i < DIM_X * DIM_Y; i++) {
        curr.board[i] = i;
    }
    curr.zero_idx = 0;
    while (curr != start) {
        Node curr_node = query(closed_set, curr);
        push_uarray(backtrack_path, curr_node.prev_action);
        curr = curr_node.prev_id;
    }

    reverse_uarray(backtrack_path);

    double kernelOverallDuration = kernelEndTime - kernelStartTime;

    printf("Kernel Duration: %f\n", kernelOverallDuration);
}

void printCudaInfo() {
    // For fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}