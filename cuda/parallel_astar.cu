#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"
#include "parallel_hash.cuh"

__device__ inline bool is_goal(int node) {
    return node == 10;
}

__device__ inline link* get_next(int node) {
    int length = 1;
    if (node > 0) {
        length = 2;
    }
    link neighbors[length];
    neighbors[0] = link(node+1, node+1);
    if (node > 0) {
        neighbors[1] = link(node-1, node);
    }
    return neighbors;
}

__device__ inline double heuristic(int node) {
    return 10 - node;
}


___global___ std::vector<int> parallel_astar_kernel(
    int start,
    int num_queues,
    int hash_table_size
) {
    if (num_queues < 1) {
        throw "We need at least one priority queue";
    }

    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Priority queue based on cost plus heuristic
    heap* pq = new_heap();

    // Hashmap for explored vertices (hashmap size should be dependent on number of nodes)
    __shared__ HashTable *closed_set = create_hash_table(hash_table_size);

    // Array S for the neighbors of the nodes on the frontier (to be deduplicated)
    __shared__ Node S[4 * num_queues];

    // Array T for the elements of S post deduplication
    __shared__ Node T[4 * num_queues];

    // Initialize data structures
    Node begin = Node(start, 0, heuristic(start));
    Node init_node_list[] = {begin};
    if (thread_idx == 0) {
        insert_deduplicate(closed_set, init_node_list, 1, pq);
    }

    // for storing the best nodes that reach the final state
    __shared__ int m = -1;
    __shared__ int fm = -1;

    // If all queues are empty, then failed to find goal state
    __shared__ bool all_empty = true;
    all_empty = all_empty && is_empty_heap(pq); // TODO: should this be ATOMIC (?)
    __syncthreads();

    while (!all_empty) {
        if (!is_empty_heap(open_set[i])) {
            element q = peak_heap(pq)[0];
            pop_heap(pq);
            int node = q.value;
            if (is_goal(node)) {
                printf("reached a goal state!\n");
                // ATOMIC REGION (?)
                int fnode = q.priority;
                if (m == -1 || fnode < fm) {
                    m = node;
                    fm = fnode;
                }
                // END ATOMIC REGION (?)
            } else {
                int cntr = 0;
                for (link neighbor : get_next(node)) {
                    assert(query(closed_set, node).g != -1);
                    
                    int cost = query(closed_set, node).g + neighbor.cost;
                    Node new_node = Node(neighbor.node, node, cost, cost+heuristic(neighbor.node));
                    
                    S[4 * thread_idx + cntr] = new_node;
                    cntr++;
                }
            }
        }

        // return best path if goal is found and there is no
        // element in any of the pq's that are less than current
        // path to goal.
        if (m != -1) {
            printf("f(m): %d\n", fm);

            __shared__ bool all_less = true;
            for (int i = 0; i < 4; i++) {
                Node n = S[i * num_queues + thread_idx];
                // unpopulated entry
                if (n.id.zero_idx == DIM_X * DIM_Y) continue;

                if (n.f < fm) {
                    all_less = false;
                    break;
                }
            }

            if (!is_empty_heap(pq) && peak_heap(pq)[0].priority < fm) {
                all_less = false;
            }

            // TODO: NEED TO PUSH ACTIONS NOT STATES - AUGMENT THE NODE STRUCT?
            if (all_less) {
                // return path to goal
                std::vector<int> backtrack_path;
                int curr = m;
                while (curr != start) {
                    backtrack_path.push_back(curr);
                    curr = query(closed_set, curr).prev_id;
                }
                backtrack_path.push_back(start);

                std::reverse(backtrack_path.begin(), backtrack_path.end());
                return backtrack_path;
            }
        }

        // deduplication section
        __shared__ bool query_mask[4 * num_queues];
        __shared__ int num_left = 4 * num_queues;
        // run in parallel
        for (size_t i = 0; i < S.size(); i++) {
            query_mask[i] = query_cost_check(closed_set, S[i]);
            if (!query_mask[i]) num_left--;
        }

        __shared__ Node *T = (Node *) malloc(num_left * sizeof(Node));
        int j = 0;
        for (size_t i = 0; i < S.size(); i++) {
            if (query_mask[i]) {
                T[j] = S[i];
                j++;
            }
        }

        // insert the remaining nodes in parallel in closed array
        // and priority queues.
        insert_deduplicate(closed_set, T, num_left, open_set);

        bool all_empty = true;
        // we can parallelize this
        for (int i = 0; i < num_queues; i++) {
            all_empty = all_empty && is_empty_heap(open_set[i]);
        }
    }

    return {};
}

void cuda_astar(int start, int num_queues, int hash_table_size) {
    // Compute number of blocks and threads per block
    // We use one thread per priority queue
    const int threadsPerBlock = 256;
    const int blocks = (num_queues + threadsPerBlock - 1) / threadsPerBlock;

    // heap<int>** device_open_set;
    // HashTable* device_closed_set;
    // Node* device_table;

    // hipMalloc((void **) &device_open_set, sizeof(heap<int>*) * num_queues);
    // hipMalloc((void **) &device_closed_set, sizeof(HashTable));
    // hipMalloc((void **) &device_table, sizeof(Node) * hash_table_size);

    double kernelStartTime = CycleTimer::currentSeconds();
    parallel_astar_kernel<<<blocks, threadsPerBlock>>>(start, num_queues, hash_table_size);
    hipDeviceSynchronize();

    double kernelEndTime = CycleTimer::currentSeconds();

    double kernelOverallDuration = kernelEndTime - kernelStartTime;

    printf("Kernel Duration: %f\n", kernelOverallDuration);
}

void printCudaInfo() {
    // For fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}