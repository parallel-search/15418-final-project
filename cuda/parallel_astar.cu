#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"
#include "parallel_hash.cuh"
#include "astar_utils.cuh"

__device__ inline bool is_goal(slider_state_t node) {
    for (unsigned char i = 0; i < DIM_X * DIM_Y; ++i) {
        if (node.board[i] != i) return false;
    }

    return true;
}

/*
__device__ inline link* get_next(slider_state_t state) {
    for (unsigned char move = UP; move <= RIGHT; ++move) {
        slider_state_t next_state = state;
        unsigned char new_zero;
        switch (move) {
            case DOWN:
                if (state.zero_idx < DIM_X) continue;
                new_zero = state.zero_idx - DIM_X;
                break;
            case UP:
                if (state.zero_idx >= DIM_X * DIM_Y - DIM_X) continue;
                new_zero = state.zero_idx + DIM_X;
                break;
            case RIGHT:
                if (state.zero_idx % DIM_X == 0) continue;
                new_zero = state.zero_idx - 1;
                break;
            case LEFT:
                if (state.zero_idx % DIM_X == DIM_X - 1) continue;
                new_zero = state.zero_idx + 1;
                break;
        }
        next_state.board[state.zero_idx] = next_state.board[new_zero];
        next_state.board[new_zero] = 0;
        next_state.zero_idx = new_zero;

        unsigned short cost = visited[state].cost + 1;
}
*/

__device__ double heuristic(slider_state_t node) {
    return 0;
}


__global__ void parallel_astar_kernel(
    slider_state_t start,
    int num_queues,
    HashTable *closed_set, 
    Node *S, 
    int *best_cost,
    int *all_queue_empty,
    int *min_goal_reached
) {
    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Priority queue based on cost plus heuristic
    heap* pq = new_heap();

    // Initialize data structures
    Node begin = Node(start, 0, heuristic(start));
    Node init_node_list[] = {begin};
    if (thread_idx == 0) {
        insert_deduplicate(closed_set, init_node_list, 1, pq);
    }

    // If all queues are empty, then failed to find goal state
    atomicAnd(all_queue_empty, is_empty_heap(pq));
    __syncthreads();

    while (!(*all_queue_empty)) {
        if (!is_empty_heap(pq)) {
            element q = peak_heap(pq)[0];
            pop_heap(pq);
            slider_state_t node = q.value;
            if (is_goal(node)) {
                printf("reached a goal state!\n");
                int fnode = q.priority;
                atomicMin(best_cost, fnode);
            } else {
                int cntr = 0;

                for (unsigned char move = UP; move <= RIGHT; ++move) {
                    unsigned short cost = query(closed_set, node).g + 1;

                    slider_state_t next_state = node;
                    unsigned char new_zero;
                    switch (move) {
                        case DOWN:
                            if (node.zero_idx < DIM_X) continue;
                            new_zero = node.zero_idx - DIM_X;
                            break;
                        case UP:
                            if (node.zero_idx >= DIM_X * DIM_Y - DIM_X) continue;
                            new_zero = node.zero_idx + DIM_X;
                            break;
                        case RIGHT:
                            if (node.zero_idx % DIM_X == 0) continue;
                            new_zero = node.zero_idx - 1;
                            break;
                        case LEFT:
                            if (node.zero_idx % DIM_X == DIM_X - 1) continue;
                            new_zero = node.zero_idx + 1;
                            break;
                    }
                    next_state.board[node.zero_idx] = next_state.board[new_zero];
                    next_state.board[new_zero] = 0;
                    next_state.zero_idx = new_zero;

                    Node new_node = Node(next_state, move, node, cost, cost+heuristic(next_state));
                    S[4 * thread_idx + cntr] = new_node;
                    cntr++;
                }
            }
        }

        // return best path if goal is found and there is no
        // element in any of the pq's that are less than current
        // path to goal.
        *min_goal_reached = true;
        __syncthreads();
        if (*best_cost != INT32_MAX) {
            for (int i = 0; i < 4; i++) {
                Node n = S[i * num_queues + thread_idx];
                // unpopulated entry
                if (n.id.zero_idx == DIM_X * DIM_Y) continue;

                atomicAnd(min_goal_reached, n.f >= *best_cost);
            }

            if (!is_empty_heap(pq) && peak_heap(pq)[0].priority < *best_cost) {
                int val = peak_heap(pq)[0].priority >= *best_cost;
                atomicAnd(min_goal_reached, val);
            }

            __syncthreads();
            if (*min_goal_reached) {
                return;
            }
        }

        // deduplication section
        // run in parallel
        for (size_t i = 0; i < 4; i++) {
            Node n = S[i * num_queues + thread_idx];
            if (query_cost_check(closed_set, n)) {
                S[i * num_queues + thread_idx].id.zero_idx = DIM_X * DIM_Y;
            }
        }
        __syncthreads();

        // insert the remaining nodes in parallel in closed array
        // and priority queues.
        for (int i = 0; i < 4; i++) {
            Node n = S[i * num_queues + thread_idx];
            if (n.id.zero_idx == DIM_X * DIM_Y) continue;

            push_heap(pq, n.id, n.f);

            int z = 0;
            int ind0 = hash_fn1(n.id, closed_set->size);
            int ind1 = hash_fn2(n.id, closed_set->size);

            if (closed_set->table[ind0].id == n.id || closed_set->table[ind0].id.zero_idx == DIM_X * DIM_Y) {
                z = 0;
            } else if (closed_set->table[ind1].id == n.id || closed_set->table[ind1].id.zero_idx == DIM_X * DIM_Y) {
                z = 1;
            }

            bool to_keep = query_cost_check(closed_set, n);
            if (z == 0 && to_keep) {
                Node old = atomicExch(closed_set->table[ind0], n);
                n = old;
            } else if (z == 1 && to_keep) {
                Node old = atomicExch(closed_set->table[ind1], n);
                n = old;    
            }
        }

        *all_queue_empty = true;
        __syncthreads();
        atomicAnd(all_queue_empty, is_empty_heap(pq));
        __syncthreads();
    }
}

void cuda_astar(slider_state_t start, int num_queues, int hash_table_size) {
    // Compute number of blocks and threads per block
    // We use one thread per priority queue
    const int threadsPerBlock = 256;
    const int blocks = (num_queues + threadsPerBlock - 1) / threadsPerBlock;

    // HashTable* closed_set = create_hash_table(hash_table_size);
    Node S[4 * num_queues];
    int best_cost = INT32_MAX;

    HashTable* device_closed_set;
    Node* device_table;
    Node* device_S;
    int* device_best_cost;
    int* device_all_queue_empty;
    int* device_min_goal_reached;

    hipMalloc((void **) &device_closed_set, sizeof(HashTable));
    hipMalloc((void **) &device_table, sizeof(Node) * hash_table_size);
    hipMalloc((void **) &device_S, sizeof(Node) * 4 * num_queues);
    hipMalloc((void **) &device_best_cost, sizeof(int));
    hipMalloc((void **) &device_all_queue_empty, sizeof(int));
    hipMalloc((void **) &device_min_goal_reached, sizeof(int));

    // hipMemcpy(device_closed_set, closed_set, sizeof(HashTable), hipMemcpyHostToDevice);
    device_closed_set = create_hash_table(hash_table_size);
    // hipMemcpy(device_table, closed_set->table, sizeof(Node) * hash_table_size, hipMemcpyHostToDevice);
    device_table = device_closed_set->table;
    hipMemcpy(device_S, S, sizeof(Node) * 4 * num_queues, hipMemcpyHostToDevice);
    hipMemcpy(device_best_cost, &best_cost, sizeof(int), hipMemcpyHostToDevice);
    *device_all_queue_empty = true;
    *device_min_goal_reached = true;

    double kernelStartTime = CycleTimer::currentSeconds();
    parallel_astar_kernel<<<blocks, threadsPerBlock>>>(start, num_queues, device_closed_set, device_S, device_best_cost, device_all_queue_empty, device_min_goal_reached);
    hipDeviceSynchronize();

    double kernelEndTime = CycleTimer::currentSeconds();

    // determine the actions using goal_state here
    uarray* backtrack_path = new_uarray(num_queues);
    slider_state_t curr;
    for (int i = 0; i < DIM_X * DIM_Y; i++) {
        curr.board[i] = i;
    }
    curr.zero_idx = 0;
    
    HashTable* closed_set;
    hipMemcpy(closed_set, device_closed_set, sizeof(HashTable), hipMemcpyDeviceToHost);
    hipMemcpy(closed_set->table, device_closed_set->table, sizeof(Node) * hash_table_size, hipMemcpyDeviceToHost);
    hipMemcpy(&best_cost, device_best_cost, sizeof(int), hipMemcpyDeviceToHost);
    while (curr != start) {
        Node curr_node = query(closed_set, curr);
        push_uarray(backtrack_path, curr_node.prev_action);
        curr = curr_node.prev_id;
    }
    reverse_uarray(backtrack_path);

    printf("Total cost: %d\n", best_cost);
    printf("Actions to goal: ");
    for (int i = 0; i < size_uarray(backtrack_path); i++) {
        printf("%d ", backtrack_path->data[i]);
    }
    printf("\n");

    double kernelOverallDuration = kernelEndTime - kernelStartTime;

    printf("Kernel Duration: %f\n", kernelOverallDuration);
}

void printCudaInfo() {
    // For fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}