#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

#define NUM_THREADS 256

bool is_goal(int node) {
    return node == 10;
}

std::vector<link<int>> get_next(int node) {
    std::vector<link<int>> neighbors;
    if (node > 0) {
        neighbors.push_back(link<int>(node-1, node));
    }
    neighbors.push_back(link<int>(node + 1, node+1));
    return neighbors;
}

double heuristic(int node) {
    return 10 - node;
}


___global___ std::vector<int> parallel_astar_kernel(
    int start,
    int num_queues,
    int hash_table_size
) {
    if (num_queues < 1) {
        throw "We need at least one priority queue";
    }

    int thread_index = blockIdx.x * blockDim.x + threadIdx.x;

    // Priority queue based on cost plus heuristic
    __shared__ heap<int>* open_set[num_queues];
    for (int i = 0; i < num_queues; i++) {
        open_set[i] = new_heap<int>();
    }

    // Hashmap for explored vertices (hashmap size should be dependent on number of nodes)
    __shared__ HashTable *closed_set = create_hash_table(hash_table_size);

    // Initialize data structures
    Node begin = Node(start, 0, heuristic(start));
    Node init_node_list[] = {begin};
    insert_deduplicate(closed_set, init_node_list, 1, open_set);

    int m = -1;
    int fm = -1;

    // If all queues are empty, then failed to find goal state
    bool all_empty = true;
    // we can parallelize this
    for (int i = 0; i < num_queues; i++) {
        all_empty = all_empty && is_empty_heap(open_set[i]);
    }

    while (!all_empty) {
        std::vector<Node> S;

        // for loop in parallel
        for (int i = 0; i < num_queues; i++) {
            // only evaluate this for loop if thread corresponding to queue index
            if (i != thread_index) continue;

            print_heap(open_set[i], i);

            if (is_empty_heap(open_set[i])) continue;

            element<int> q = peak_heap(open_set[i])[0];
            pop_heap(open_set[i]);
            int node = q.value;
            if (is_goal(node)) {
                printf("reached a goal state!\n");
                // ATOMIC REGION (?)
                int fnode = q.priority;
                if (m == -1 || fnode < fm) {
                    m = node;
                    fm = fnode;
                }
                // END ATOMIC REGION (?)
                continue;
            }

            for (link<int> neighbor : get_next(node)) {
                assert(query(closed_set, node).g != -1);
                int cost = query(closed_set, node).g + neighbor.cost;
                Node new_node = Node(neighbor.node, node, cost, cost+heuristic(neighbor.node));
                S.push_back(new_node);
            }
        }

        // return best path if goal is found and there is no
        // element in any of the pq's that are less than current
        // path to goal.
        if (m != -1) {
            printf("f(m): %d\n", fm);
            bool all_less = true;

            for (Node n : S) {
                if (n.f < fm) {
                    all_less = false;
                    break;
                }
            }

            for (int i = 0; i < num_queues; i++) {
                if (is_empty_heap(open_set[i])) continue;

                int min_cost = peak_heap(open_set[i])[0].priority;
                if (min_cost < fm) {
                    all_less = false;
                    break;
                }
            }

            if (all_less) {
                // return path to goal
                std::vector<int> backtrack_path;
                int curr = m;
                while (curr != start) {
                    backtrack_path.push_back(curr);
                    curr = query(closed_set, curr).prev_id;
                }
                backtrack_path.push_back(start);

                std::reverse(backtrack_path.begin(), backtrack_path.end());
                return backtrack_path;
            }
        }

        // deduplication section
        __shared__ bool query_mask[S.size()];
        __shared__ int num_left = S.size();
        // run in parallel
        for (size_t i = 0; i < S.size(); i++) {
            query_mask[i] = query_cost_check(closed_set, S[i]);
            if (!query_mask[i]) num_left--;
        }

        __shared__ Node *T = (Node *) malloc(num_left * sizeof(Node));
        int j = 0;
        for (size_t i = 0; i < S.size(); i++) {
            if (query_mask[i]) {
                T[j] = S[i];
                j++;
            }
        }

        // insert the remaining nodes in parallel in closed array
        // and priority queues.
        insert_deduplicate(closed_set, T, num_left, open_set);

        bool all_empty = true;
        // we can parallelize this
        for (int i = 0; i < num_queues; i++) {
            all_empty = all_empty && is_empty_heap(open_set[i]);
        }
    }

    return {};
}

void cuda_astar(int start, int num_queues, int hash_table_size) {
    // Compute number of blocks and threads per block
    // We use one thread per priority queue
    const int threadsPerBlock = NUM_THREADS;
    const int blocks = (num_queues + threadsPerBlock - 1) / threadsPerBlock;

    double kernelStartTime = CycleTimer::currentSeconds();
    parallel_astar_kernel<<<blocks, threadsPerBlock>>>(start, num_queues, hash_table_size);
    hipDeviceSynchronize();

    double kernelEndTime = CycleTimer::currentSeconds();

    double kernelOverallDuration = kernelEndTime - kernelStartTime;

    printf("Kernel Duration: %f\n", kernelOverallDuration);
}

void printCudaInfo() {
    // For fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}